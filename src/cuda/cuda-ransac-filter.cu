
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-ransac-filter.cuh"
#include <iostream>
#include "../../include/librealsense2/rsutil.h"

void depth_to_3d_space(
    librealsense::float3 *point, 
    const rs2_intrinsics &depth_intrinsics, 
    int location,
    float depth)
{
    // Convert location to x, y coordinates.
    int width = depth_intrinsics.width;
    int height = depth_intrinsics.height;
    const float pixel[] = { ((float)location % width), ((float)location / width) };
    librealsense::rs2_deproject_pixel_to_point(point, depth_intrinsics, pixel, depth);
}

/**
 * Unlike with the serial version, we directly use the depth data (not points) as
 * we want to avoid the overhead of copying the points from the device to host.
 **/
void rscuda::generate_equation(
    const uint16_t *depth_data, 
    librealsense::float4 *equation, 
    int size,
    const rs2_intrinsics &depth_intrinsics, 
    float depth_scale)
{
    librealsense::float3 vector_ab = {};
    librealsense::float3 vector_ac = {};
    librealsense::float3 point_a = {};
    // Ensure that the 3 points we're using for the plane generation are 
    // not colinnear and their depths are not 0.
    bool invalid = true;
    do { 
	// Select 3 unique random points.
	int a = 0, b = 0, c = 0;
	while (a == b || b == c || a == c) {
	    a = rand() % (size-1);
	    b = rand() % (size-1);
	    c = rand() % (size-1);
	}

        // For each of our points, get their location in 3d space.
        depth_to_3d_space(&point_a, depth_intrinsics, a, depth_scale * depth_data[a])
	float3 point_b = {};
        depth_to_3d_space(&point_b, depth_intrinsics, b, depth_scale * depth_data[b])
	float3 point_c = {};
        depth_to_3d_space(&point_c, depth_intrinsics, c, depth_scale * depth_data[c])

	// Don't use holes in the image with depth of 0.0.
	if (point_a.z < 0.01f) continue;
	if (point_b.z < 0.01f) continue;
	if (point_c.z < 0.01f) continue;

	// Compute our vectors.
	vector_ab = point_b - point_a;
	vector_ac = point_c - point_a;

	// Ensure that the three points are not collinear by ensuring the 
	// vectors are not parallel. Parallel vectors will have a similar ratio
	// beween their components. Ex: vectora = 4 * vectorb (they are parallel)
	float ratio_x = vector_ab.x / vector_ac.x;
	float ratio_y = vector_ab.y / vector_ac.y;
	float ratio_z = vector_ab.z / vector_ac.z;
	bool equal_xy = fabs(ratio_x - ratio_y) < 0.01;
	bool equal_yz = fabs(ratio_y - ratio_z) < 0.01;
	bool equal_xz = fabs(ratio_x - ratio_z) < 0.01;
	invalid = equal_xy && equal_yz && equal_xz;
    } while(invalid);

    // Compute the cross product of the vectors.
    float cpx = vector_ab.y * vector_ac.z - vector_ab.z * vector_ac.y;
    float cpy = vector_ab.z * vector_ac.x - vector_ab.x * vector_ac.z;
    float cpz = vector_ab.x * vector_ac.y - vector_ab.y * vector_ac.x;

    // Use the cross product and point a to find the constant in the plane equation.
    float d = -(cpx * point_a.x + cpy * point_a.y + cpz * point_a.z);
    equation->x = cpx;
    equation->y = cpy;
    equation->x = cpz; 
}

__global__
void rscuda::get_inliers(

void rscuda::ransac_filter_cuda(
    bool *inliers, 
    const uint16_t * depth_data, 
    librealsense::float3 *points, 
    int size, 
    const rs2_intrinsics &depth_intrinsics, 
    float *depth_scale, 
    bool &plane_found, 
    librealsense::float4 &equation, 
    const float distance_threshold,
    const float threshold_percent,
    const float iterations)
{
    // Initialize random seed.
    srand((unsigned)time(0));

    // RANSAC settings.
    int inlier_threshold_count = (((int)threshold_percent) * size) / 100;

    // CUDA KERNEL DEPTH_TO_POINTS
    int count = depth_intrinsics.height * depth_intrinsics.width;
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;

    // Declare device variables.
    float *dev_points = 0;
    uint16_t *dev_depth_data = 0;
    rs2_intrinsics* dev_intrin = 0;
    float *dev_depth_scale = 0;
    librealsense::float4 *dev_equation = 0;
    int *dev_size = 0;
    bool *dev_inliers = 0;
    float *dev_distance_threshold = 0;

    cudaError_t result;

    // Allocate Memory on the Device.
    result = cudaMalloc(&dev_points, count * sizeof(float) * 3);
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_depth_data, count * sizeof(uint16_t));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_intrin, sizeof(rs2_intrinsics));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_depth_scale, sizeof(float));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_equation, sizeof(librealsense::float4));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_size, sizeof(int));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_inliers, count * sizeof(bool));
    assert(result == cudaSuccess);
    result = cudaMalloc(&dev_distance_threshold, sizeof(float));
    assert(result == cudaSuccess);

    // Copy values over to cuda device.
    result = cudaMemcpy(dev_depth, depth_data, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess); 
    result = cudaMemcpy(dev_intrin, &intrin, sizeof(rs2_intrinsics), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess); 
    result = cudaMemcpy(dev_depth_scale, &depth_scale, sizeof(float), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess); 

 
    kernel_deproject_depth_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(dev_points, dev_intrin, dev_depth_data, dev_depth_scale);     
    cudaDeviceSynchronize();

    for (int j = 0; j < (int)iterations; j++) {
        // Generate a random plane equation, if our last equation did not find a plane.
	if (!plane_found) {
            generate_equation(depth_data, equation, size, depth_intrinsics, depth_scale);
	}

        // Get the inliers & count using this equation.
	get_inliers<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(dev_equation, dev_points, dev_size, dev_inliers, dev_distance_threshold);
        cudaDeviceSynchronize();

        //copy inliers from device to host



        // loop through array of inliers
        int inlier_count = 0;
        for (int i = 0; i < size; i++) {

        }
	if (inlier_count >= inlier_threshold_count) {
            plane_found = true;
            break;
        }
        else {
            plane_found = false;
	}
    }
}

#endif
